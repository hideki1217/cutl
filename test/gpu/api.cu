#include "hip/hip_runtime.h"
#include "./api.hpp"

#include <algorithm>
#include <hip/hip_runtime_api.h>

#include "../util.hpp"
#include "cutl.cuh"

namespace gpu{

template<int N>
class Collatz {
public:
    __device__ __host__ inline
    int operator() (int x) const {
        for(int i=0; i<N; i++) {
            x = (x%2 == 0) ? x/2 : 3*x+1;
        }
        return x;
    }
};

CollatzMap_1024::CollatzMap_1024()  { hipMalloc(&block, sizeof(int) * BlockN); }
CollatzMap_1024::~CollatzMap_1024() { hipFree(block); }
void CollatzMap_1024::call(const int* in, int* out, int n) {
        for(int i=0; i<n; i+=BlockN){
            hipMemcpy(block, in + i, sizeof(int) * BlockN, hipMemcpyHostToDevice);
            cutl::map_unroll<2><<<1, BlockN/2>>>(Collatz<BlockN>(), block, block, min(BlockN, n - i));
            hipMemcpy(out + i, block, sizeof(int) * min(BlockN, n - i), hipMemcpyDeviceToHost);
        }
    }
}